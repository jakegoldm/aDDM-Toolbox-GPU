#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../include/ddm.cuh"
#include "../include/util.h"
#include "../include/cuda_util.cuh"


__global__
void getTrialLikelihoodKernel(
    bool debug, 
    int trialsPerThread, 
    int *RTs, 
    int *choices, 
    int *valDiffs, 
    double* likelihoods,
    int numTrials, 
    float *states, 
    int biasState,
    int numStates, 
    float stateStep, 
    float d, 
    float sigma, 
    int barrier, 
    int nonDecisionTime, 
    int timeStep, 
    float approxStateStep, 
    float dec) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < numTrials / trialsPerThread) {
        for (int trialNum = tid * trialsPerThread; trialNum < (tid + 1) * trialsPerThread; trialNum++) {

            int choice = choices[trialNum];
            int RT = RTs[trialNum];
            int valDiff = valDiffs[trialNum];


            int numTimeSteps = RT / timeStep; 

            // requires compute capability 2.x
            float* barrierUp = new float[numTimeSteps];
            float *barrierDown = new float[numTimeSteps];

            for (int i = 0 ; i < numTimeSteps; i++) {
                barrierUp[i] = barrier / (1 + (dec * i));
                barrierDown[i] = -barrier / (1 + (dec * i));
            }



            double *prStates = new double[numStates];
            for (int i = 0; i < numStates; i++) {
                prStates[i] = (i == biasState) ? 1 : 0; 
            }

            double *probUpCrossing = new double[numTimeSteps];
            double *probDownCrossing = new double[numTimeSteps];
            for (int i = 0; i < numTimeSteps; i++) {
                probUpCrossing[i] = 0; 
                probDownCrossing[i] = 0; 
            }

            if (debug) {
                for (int i = 0 ; i < numStates ; i++) {
                    printf("prStates[%i] = %f\n", i, prStates[i]);
                }
            }
            

            float *changeMatrix = new float[numStates * numStates];
            for (int i = 0; i < numStates; i++) {
                for (int j = 0; j < numStates; j++) {
                    changeMatrix[__RC2IDX(i, j, numStates)] = states[i] - states[j];
                }
            }

            float *changeUp = new float[numStates * numTimeSteps];
            for (int i = 0; i < numStates; i++) {
                for (int j = 0; j < numTimeSteps; j++) {
                    changeUp[__RC2IDX(i, j, numTimeSteps)] = barrierUp[j] - states[i];
                }
            }

            float *changeDown = new float[numStates * numTimeSteps];
            for (int i = 0; i < numStates; i++) {
                for (int j = 0; j < numTimeSteps; j++) {
                    changeDown[__RC2IDX(i, j, numTimeSteps)] = barrierDown[j] - states[i];
                }
            }

            if (debug) {
                printf("change matrix\n");
                for (int i = 0; i < numStates * numStates; i++) {
                    printf("%f ", changeMatrix[i]);
                    if ((i + 1) % numStates == 0) {
                        printf("\n");
                    }
                }

                printf("change up\n");
                for (int i = 0; i < numStates * numTimeSteps; i++) {
                    printf("%f ", changeUp[i]);
                    if ((i + 1) % numTimeSteps == 0) {
                        printf("\n");
                    }
                }
            }


            int elapsedNDT = 0;
            bool recomputePDCM = true; 
            float prevMean = 0; 
            float *probDistChangeMatrix = new float[numStates * numStates];
            double* prStatesNew = new double[numStates];
            float *changeUpCDFs = new float[numStates];
            float *changeDownCDFs = new float[numStates];

            for (int time = 1; time < numTimeSteps; time++) {

                if (debug) printf(
                    "============\n timestep %i \n============", time
                );

                float mean; 
                if (elapsedNDT < nonDecisionTime / timeStep) {
                    mean = 0; 
                    elapsedNDT += 1; 
                } else {
                    mean = d * valDiff;
                }

                if (mean != prevMean) {
                    recomputePDCM = true;
                } else {
                    recomputePDCM = false; 
                }

                if (recomputePDCM || time == 1) {
                    for (int i = 0; i < numStates; i++) {
                        for (int j = 0; j < numStates; j++) {
                            float x = changeMatrix[__RC2IDX(i, j, numStates)];
                            probDistChangeMatrix[__RC2IDX(i, j, numStates)] = pdf(x, mean, sigma);
                        }
                    }
                }

                if (debug) {
                    printf("PDCM\n");
                    for (int i = 0; i < numStates * numStates; i++) {
                        printf("%f ", probDistChangeMatrix[i]);
                        if ((i + 1) % numStates == 0) {
                            printf("\n");
                        }
                    }
                }

                double rowSum; 
                for (int i = 0; i < numStates; i++) {
                    rowSum = 0; 
                    for (int j = 0; j < numStates; j++) {
                        rowSum += stateStep * probDistChangeMatrix[__RC2IDX(i, j, numStates)] * prStates[j];
                    }
                    prStatesNew[i] = (states[i] > barrierUp[time] || states[i] < barrierDown[time]) ? 0 : rowSum;
                }

                if (debug) {
                    for (int i = 0 ; i < numStates ; i++) {
                        printf("prStatesNew[%i] = %f\n", i, prStatesNew[i]);
                    }
                }

                for (int i = 0; i < numStates; i++) {
                    float x = changeUp[__RC2IDX(i, time, numTimeSteps)];
                    changeUpCDFs[i] = 1 - normcdff((x - mean) / sigma);
                }
                if (debug) {
                    for (int i = 0; i < numStates; i++) {
                        printf("changeUpCDFs[%i] = %f\n", i, changeUpCDFs[i]);
                    }
                }
                double tempUpCross = 0; 
                for (int i = 0; i < numStates; i++) {
                    tempUpCross += changeUpCDFs[i] * prStates[i];
                }

                for (int i = 0; i < numStates; i++) {
                    float x = changeDown[__RC2IDX(i, time, numTimeSteps)];
                    changeDownCDFs[i] = normcdff((x - mean) / sigma);
                }
                if (debug) {
                    for (int i = 0; i < numStates; i++) {
                        printf("changeDownCDFs[%i] = %f\n", i, changeDownCDFs[i]);
                    }
                }
                double tempDownCross = 0; 
                for (int i = 0; i < numStates; i++) {
                    tempDownCross += changeDownCDFs[i] * prStates[i];
                }

                if (debug) printf("temp up cross = %f\n", tempUpCross);
                if (debug) printf("temp down cross = %f\n", tempDownCross);

                double sumIn = 0; 
                double sumCurrent = tempUpCross + tempDownCross; 
                for (int i = 0; i < numStates; i++) {
                    sumIn += prStates[i];
                    sumCurrent += prStatesNew[i];
                }
                double normFactor = sumIn / sumCurrent; 
                for (int i = 0; i < numStates; i++) {
                    prStates[i] = prStatesNew[i] * normFactor; 
                }

                probUpCrossing[time] = tempUpCross * normFactor; 
                probDownCrossing[time] = tempDownCross * normFactor;

                prevMean = mean;
            }

            double likelihood = 0; 
            if (choice == -1) {
                if (probUpCrossing[numTimeSteps - 1] > 0) {
                    likelihood = probUpCrossing[numTimeSteps - 1];
                }
            } else if (choice == 1) {
                if (probDownCrossing[numTimeSteps - 1] > 0) {
                    likelihood = probDownCrossing[numTimeSteps - 1];
                }
            }

            delete[] barrierUp;
            delete[] barrierDown;
            delete[] probUpCrossing;
            delete[] probDownCrossing;
            delete[] prStates;
            delete[] changeMatrix;
            delete[] changeUp;
            delete[] changeDown;
            delete[] probDistChangeMatrix;
            delete[] prStatesNew;
            delete[] changeUpCDFs;
            delete[] changeDownCDFs;

            if (likelihood == 0) {
                likelihood = pow(10, -20);
            }            
            likelihoods[trialNum] = -log(likelihood);
        }
    }    
}

void DDM::callGetTrialLikelihoodKernel(
    bool debug, int trialsPerThread, int numBlocks, int threadsPerBlock, 
    DDMTrial *trials, double *likelihoods, 
    int numTrials, float d, float sigma, float barrier, 
    int nonDecisionTime, int timeStep, float approxStateStep, float dec) {

    int *d_RTs, *d_choices, *d_VDs;
    hipMalloc((void**)&d_RTs, numTrials * sizeof(int));
    hipMalloc((void**)&d_choices, numTrials * sizeof(int));
    hipMalloc((void**)&d_VDs, numTrials * sizeof(int));

    int *h_VDs = new int[numTrials];
    int *h_RTs = new int[numTrials];
    int *h_choices = new int[numTrials];
    for (int i = 0; i < numTrials; i++) {
        h_VDs[i] = trials[i].valueLeft - trials[i].valueRight;
        h_RTs[i] = trials[i].RT;
        h_choices[i] = trials[i].choice;
    }

    hipMemcpy(d_RTs, h_RTs, numTrials * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_choices, h_choices, numTrials * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_VDs, h_VDs, numTrials * sizeof(int), hipMemcpyHostToDevice);
    
    int halfNumStateBins = ceil(barrier / approxStateStep); 
    if (debug) printf("half num state bins %i\n", halfNumStateBins);
    float stateStep = barrier / (halfNumStateBins + 0.5);
    if (debug) printf("state step %f\n", stateStep);
    int numStates = 2 * halfNumStateBins + 1; 

    float *states = new float[numStates];
    int s_idx = 0; 
    float biasStateVal = MAXFLOAT; 
    int biasState; 
    float r; 
    
    for (float ss = -barrier + (stateStep / 2); ss <= barrier - (stateStep / 2); ss += stateStep) {
        states[s_idx] = ss;
        r = abs(ss - bias); 
        if (r < biasStateVal) {
            biasState = s_idx;
            biasStateVal = r; 
        }
        s_idx++;
    }

    float *d_states; 
    hipMalloc((void**) &d_states, numStates * sizeof(float));
    hipMemcpy(d_states, states, numStates * sizeof(float), hipMemcpyHostToDevice);

    getTrialLikelihoodKernel<<<numBlocks, threadsPerBlock>>>(
        debug,
        trialsPerThread,
        d_RTs,
        d_choices,
        d_VDs,
        likelihoods,
        numTrials,
        d_states, 
        biasState,
        numStates,
        stateStep,
        d, sigma, barrier,
        nonDecisionTime,
        timeStep,
        approxStateStep,
        dec
    );

    hipFree(d_RTs);
    hipFree(d_choices);
    hipFree(d_VDs);
    hipFree(d_states);
    delete[] h_RTs;
    delete[] h_choices;
    delete[] h_VDs;
    delete[] states;
    }
        

double DDM::computeGPUNLL(std::vector<DDMTrial> trials, bool debug, int trialsPerThread, int timeStep, float approxStateStep) {
    int numTrials = trials.size(); 

    DDMTrial *d_trials;
    double *d_likelihoods;
    hipMalloc((void**) &d_trials, numTrials * sizeof(DDMTrial));
    hipMalloc((void**) &d_likelihoods, numTrials * sizeof(double));
    hipMemcpy(d_trials, trials.data(), numTrials * sizeof(DDMTrial), hipMemcpyHostToDevice);

    int threadsPerBlock = 256; 
    int numBlocks = 16;

    DDM::callGetTrialLikelihoodKernel(
        debug, trialsPerThread, numBlocks, threadsPerBlock, 
        trials.data(), d_likelihoods, 
        numTrials, d, sigma, barrier, 
        nonDecisionTime, timeStep, approxStateStep, DECAY);

    std::vector<double> h_likelihoods(numTrials);
    hipMemcpy(h_likelihoods.data(), d_likelihoods, numTrials * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_trials);
    hipFree(d_likelihoods);

    double NLL = 0;
    for (int i = 0; i < numTrials; i++) {
        NLL += h_likelihoods[i];
    }

    return NLL;
}

DDM DDM::fitModelMLE(std::vector<DDMTrial> trials, std::vector<float> rangeD, std::vector<float> rangeSigma, float barrier, std::string computeMethod) {
    if (std::find(validComputeMethods.begin(), validComputeMethods.end(), computeMethod) == validComputeMethods.end()) {
        throw std::invalid_argument("Input computeMethod is invalid.");
    } 
    sort(rangeD.begin(), rangeD.end());
    sort(rangeSigma.begin(), rangeSigma.end());

    std::vector<DDM> potentialModels; 
    for (float d : rangeD) {
        for (float sigma : rangeSigma) {
            DDM ddm = DDM(d, sigma, barrier);
            potentialModels.push_back(ddm);
        }
    }

    std::function<double(DDM)> NLLcomputer; 
    if (computeMethod == "basic") {
        NLLcomputer = [trials](DDM ddm) -> double {
            double NLL = 0; 
            for (DDMTrial trial : trials) {
                NLL += -log(ddm.getTrialLikelihood(trial));
            }
            return NLL;
        };
    } 
    else if (computeMethod == "thread") {
        NLLcomputer = [trials](DDM ddm) -> double {
            return ddm.computeParallelNLL(trials);
        };
    } 
    else if (computeMethod == "gpu") {
        NLLcomputer = [trials](DDM ddm) -> double {
            return ddm.computeGPUNLL(trials);
        };
    }   
    
    double minNLL = __DBL_MAX__;
    DDM optimal = DDM(); 
    for (DDM ddm : potentialModels) {
        double NLL = NLLcomputer(ddm);
        std::cout << "testing d=" << ddm.d << " sigma=" << ddm.sigma << " NLL=" << NLL << std::endl; 
        if (NLL < minNLL) {
            minNLL = NLL; 
            optimal = ddm; 
        }
    }
    return optimal; 
}
