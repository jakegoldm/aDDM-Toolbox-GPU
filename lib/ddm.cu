#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../include/ddm.cuh"
#include "../include/util.h"

__device__ int __RC2IDX(int row, int col, int columns_per_row) {
    return (row * columns_per_row) + col; 
}

__device__ double pdf(float x, float mean, float sigma) {
    double first = exp(-0.5 * pow((x - mean) / sigma, 2));
    double second = sigma * sqrt(2 * M_PI);
    return first / second; 
}

__global__
void getTrialLikelihoodKernel(bool debug, int trialsPerThread, int *RTs, int *choices, int *valDiffs, double* likelihoods, int numTrials, float d, float sigma, int barrier, int nonDecisionTime, int timeStep, float approxStateStep, float dec) {
int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // printf("TID = %i\n", tid);

    if (tid < numTrials / trialsPerThread) {

        for (int trialNum = tid * trialsPerThread; trialNum < (tid + 1) * trialsPerThread; trialNum++) {

    

        // printf("entering for tid %i\n", tid);
        int choice = choices[trialNum];
        int RT = RTs[trialNum];
        int valDiff = valDiffs[trialNum];


        // printf("choice %i, RT %i, vd %i\n", choice, RT, valDiff);

        float bias = 0; 

        int numTimeSteps = RT / timeStep; 

        // requires compute capability 2.x
        float* barrierUp = new float[numTimeSteps];
        float *barrierDown = new float[numTimeSteps];

        for (int i = 0 ; i < numTimeSteps; i++) {
            barrierUp[i] = barrier / (1 + (dec * i));
            barrierDown[i] = -barrier / (1 + (dec * i));
        }

        int halfNumStateBins = ceil(barrier / approxStateStep); 
        if (debug) printf("half num state bins %i\n", halfNumStateBins);
        float stateStep = barrier / (halfNumStateBins + 0.5);
        if (debug) printf("state step %f\n", stateStep);
        int numStates = 2 * halfNumStateBins + 1; 


        float *states = new float[numStates];
        int s = 0; 
        float biasStateVal = MAXFLOAT; 
        int biasState; 
        float r; 
        for (float ss = barrierDown[0] + (stateStep / 2); ss <= barrierUp[0] - (stateStep / 2); ss += stateStep) {
            states[s] = ss;
            r = abs(ss - bias); 
            if (r < biasStateVal) {
                biasState = s;
                biasStateVal = r; 
            }
            s++;
        }

        if (debug) {
            for (int i = 0; i < numStates; i++) {
                printf("states[%i] = %f\n", i, states[i]);
            }
            printf("bias state %i\n", biasState);
        }

        double *prStates = new double[numStates];
        for (int i = 0; i < numStates; i++) {
            prStates[i] = (i == biasState) ? 1 : 0; 
        }

        double *probUpCrossing = new double[numTimeSteps];
        double *probDownCrossing = new double[numTimeSteps];
        for (int i = 0; i < numTimeSteps; i++) {
            probUpCrossing[i] = 0; 
            probDownCrossing[i] = 0; 
        }


        if (debug) {
            for (int i = 0 ; i < numStates ; i++) {
                printf("prStates[%i] = %f\n", i, prStates[i]);
            }
        }
        

        float *changeMatrix = new float[numStates * numStates];
        for (int i = 0; i < numStates; i++) {
            for (int j = 0; j < numStates; j++) {
                changeMatrix[__RC2IDX(i, j, numStates)] = states[i] - states[j];
            }
        }

        float *changeUp = new float[numStates * numTimeSteps];
        for (int i = 0; i < numStates; i++) {
            for (int j = 0; j < numTimeSteps; j++) {
                changeUp[__RC2IDX(i, j, numTimeSteps)] = barrierUp[j] - states[i];
            }
        }

        float *changeDown = new float[numStates * numTimeSteps];
        for (int i = 0; i < numStates; i++) {
            for (int j = 0; j < numTimeSteps; j++) {
                changeDown[__RC2IDX(i, j, numTimeSteps)] = barrierDown[j] - states[i];
            }
        }

        if (debug) {
            printf("change matrix\n");
            for (int i = 0; i < numStates * numStates; i++) {
                printf("%f ", changeMatrix[i]);
                if ((i + 1) % numStates == 0) {
                    printf("\n");
                }
            }

            printf("change up\n");
            for (int i = 0; i < numStates * numTimeSteps; i++) {
                printf("%f ", changeUp[i]);
                if ((i + 1) % numTimeSteps == 0) {
                    printf("\n");
                }
            }
        }


        int elapsedNDT = 0;
        bool recomputePDCM = true; 
        float prevMean = 0; 
        float *probDistChangeMatrix = new float[numStates * numStates];

        for (int time = 1; time < numTimeSteps; time++) {

            if (debug) printf(
                "============\n timestep %i \n============", time
            );

            float mean; 
            if (elapsedNDT < nonDecisionTime / timeStep) {
                mean = 0; 
                elapsedNDT += 1; 
            } else {
                mean = d * valDiff;
            }

            if (mean != prevMean) {
                recomputePDCM = true;
            } else {
                recomputePDCM = false; 
            }

            if (recomputePDCM || time == 1) {
                for (int i = 0; i < numStates; i++) {
                    for (int j = 0; j < numStates; j++) {
                        float x = changeMatrix[__RC2IDX(i, j, numStates)];
                        probDistChangeMatrix[__RC2IDX(i, j, numStates)] = pdf(x, mean, sigma);
                    }
                }
            }

            if (debug) {
                printf("PDCM\n");
                for (int i = 0; i < numStates * numStates; i++) {
                    printf("%f ", probDistChangeMatrix[i]);
                    if ((i + 1) % numStates == 0) {
                        printf("\n");
                    }
                }
            }

            double rowSum; 
            double* prStatesNew = new double[numStates];
            for (int i = 0; i < numStates; i++) {
                rowSum = 0; 
                for (int j = 0; j < numStates; j++) {
                    rowSum += stateStep * probDistChangeMatrix[__RC2IDX(i, j, numStates)] * prStates[j];
                }
                prStatesNew[i] = (states[i] > barrierUp[time] || states[i] < barrierDown[time]) ? 0 : rowSum;
            }

            if (debug) {
                for (int i = 0 ; i < numStates ; i++) {
                    printf("prStatesNew[%i] = %f\n", i, prStatesNew[i]);
                }
            }

            float *changeUpCDFs = new float[numStates];
            for (int i = 0; i < numStates; i++) {
                float x = changeUp[__RC2IDX(i, time, numTimeSteps)];
                changeUpCDFs[i] = 1 - normcdf((x - mean) / sigma);
            }
            if (debug) {
                for (int i = 0; i < numStates; i++) {
                    printf("changeUpCDFs[%i] = %f\n", i, changeUpCDFs[i]);
                }
            }
            double tempUpCross = 0; 
            for (int i = 0; i < numStates; i++) {
                tempUpCross += changeUpCDFs[i] * prStates[i];
            }

            float *changeDownCDFs = new float[numStates];
            for (int i = 0; i < numStates; i++) {
                float x = changeDown[__RC2IDX(i, time, numTimeSteps)];
                changeDownCDFs[i] = normcdf((x - mean) / sigma);
            }
            if (debug) {
                for (int i = 0; i < numStates; i++) {
                    printf("changeDownCDFs[%i] = %f\n", i, changeDownCDFs[i]);
                }
            }
            double tempDownCross = 0; 
            for (int i = 0; i < numStates; i++) {
                tempDownCross += changeDownCDFs[i] * prStates[i];
            }

            if (debug) printf("temp up cross = %f\n", tempUpCross);
            if (debug) printf("temp down cross = %f\n", tempDownCross);

            double sumIn = 0; 
            double sumCurrent = tempUpCross + tempDownCross; 
            for (int i = 0; i < numStates; i++) {
                sumIn += prStates[i];
                sumCurrent += prStatesNew[i];
            }
            double normFactor = sumIn / sumCurrent; 
            for (int i = 0; i < numStates; i++) {
                prStates[i] = prStatesNew[i] * normFactor; 
            }

            probUpCrossing[time] = tempUpCross * normFactor; 
            probDownCrossing[time] = tempDownCross * normFactor;

            prevMean = mean;

            delete[] prStatesNew;
            delete[] changeUpCDFs;
            delete[] changeDownCDFs;
        }

        double likelihood = 0; 
        if (choice == -1) {
            if (probUpCrossing[numTimeSteps - 1] > 0) {
                likelihood = probUpCrossing[numTimeSteps - 1];
            }
        } else if (choice == 1) {
            if (probDownCrossing[numTimeSteps - 1] > 0) {
                likelihood = probDownCrossing[numTimeSteps - 1];
            }
        }

        delete[] barrierUp;
        delete[] barrierDown;
        delete[] probUpCrossing;
        delete[] probDownCrossing;
        delete[] states;
        delete[] prStates;
        delete[] changeMatrix;
        delete[] changeUp;
        delete[] changeDown;
        delete[] probDistChangeMatrix;
        
        likelihoods[trialNum] = -log(likelihood);
        }
    }    
}

void DDM::callGetTrialLikelihoodKernel(
    bool debug, int trialsPerThread, int numBlocks, int threadsPerBlock, 
    DDMTrial *trials, double *likelihoods, 
    int numTrials, float d, float sigma, float barrier, 
    int nonDecisionTime, int timeStep, float approxStateStep, float dec) {

    int *d_RTs, *d_choices, *d_VDs;
    hipMalloc((void**)&d_RTs, numTrials * sizeof(int));
    hipMalloc((void**)&d_choices, numTrials * sizeof(int));
    hipMalloc((void**)&d_VDs, numTrials * sizeof(int));

    int *h_VDs = new int[numTrials];
    int *h_RTs = new int[numTrials];
    int *h_choices = new int[numTrials];
    for (int i = 0; i < numTrials; i++) {
        h_VDs[i] = trials[i].valueLeft - trials[i].valueRight;
        h_RTs[i] = trials[i].RT;
        h_choices[i] = trials[i].choice;
    }

    hipMemcpy(d_RTs, h_RTs, numTrials * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_choices, h_choices, numTrials * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_VDs, h_VDs, numTrials * sizeof(int), hipMemcpyHostToDevice);

    getTrialLikelihoodKernel<<<numBlocks, threadsPerBlock>>>(
        debug,
        trialsPerThread,
        d_RTs,
        d_choices,
        d_VDs,
        likelihoods,
        numTrials,
        d, sigma, barrier,
        nonDecisionTime,
        timeStep,
        approxStateStep,
        dec
    );

    hipFree(d_RTs);
    hipFree(d_choices);
    hipFree(d_VDs);
    delete[] h_RTs;
    delete[] h_choices;
    delete[] h_VDs;
    }
        

double DDM::computeGPUNLL(std::vector<DDMTrial> trials, bool debug, int trialsPerThread, int timeStep, float approxStateStep) {
    int numTrials = trials.size(); 

    DDMTrial* d_trials;
    double* d_likelihoods;
    hipMalloc((void**) &d_trials, numTrials * sizeof(DDMTrial));
    hipMalloc((void**) &d_likelihoods, numTrials * sizeof(double));
    hipMemcpy(d_trials, trials.data(), numTrials * sizeof(DDMTrial), hipMemcpyHostToDevice);

    int threadsPerBlock = 64; 
    int numBlocks = 16;

    callGetTrialLikelihoodKernel(
        debug, trialsPerThread, numBlocks, threadsPerBlock, 
        trials.data(), d_likelihoods, 
        numTrials, d, sigma, barrier, 
        nonDecisionTime, timeStep, approxStateStep, DECAY);

    std::vector<double> h_likelihoods(numTrials);
    hipMemcpy(h_likelihoods.data(), d_likelihoods, numTrials * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_trials);
    hipFree(d_likelihoods);

    double NLL = 0;
    for (int i = 0; i < numTrials; i++) {
        NLL += h_likelihoods[i];
    }

    return NLL;

}