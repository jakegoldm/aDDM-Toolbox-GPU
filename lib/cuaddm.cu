#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cassert>
#include "../include/addm.cuh"
#include "../include/ddm.cuh"
#include "../include/cuda_util.cuh"
#include "../include/util.h"


__global__
void getTrialLikelihoodKernel(
    bool debug, 
    int trialsPerThread, 
    int *RTs, 
    int *choices, 
    int *valLs, 
    int *valRs, 
    int *FixItemsMatrix, 
    int *FixTimeMatrix, 
    int *FixLens, 
    double *likelihoods, 
    int numTrials, 
    float *states, 
    int maxFixLen, 
    int biasState, 
    int numStates,
    float stateStep,
    float d, 
    float sigma, 
    float theta, 
    int barrier, 
    int nonDecisionTime, 
    int timeStep, 
    float approxStateStep, 
    float dec) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x; 
    if (tid < numTrials / trialsPerThread) {
        for (int trialNum = tid * trialsPerThread; trialNum < (tid + 1) * trialsPerThread; trialNum++) {
            
            int choice = choices[trialNum];
            int RT = RTs[trialNum];
            int valLeft = valLs[trialNum];
            int valRight = valRs[trialNum];
            int fixLen = FixLens[trialNum];

            int *fixItem = new int[fixLen];
            int *fixTime = new int[fixLen];

            int f_idx = 0; 
            for (int i = trialNum * maxFixLen; i < trialNum * maxFixLen + fixLen; i++) {
                fixItem[f_idx] = FixItemsMatrix[i];
                fixTime[f_idx] = FixTimeMatrix[i];
                f_idx++; 
            }

            if (debug) {
                printf("%i %i %i %i\n", choice, RT, valLeft - valRight, fixLen);
                printf("Fix Item | Fix Time \n");
                for (int i = 0; i < fixLen; i++) {
                    printf("%i        | %i   \n", fixItem[i], fixTime[i]);
                } 
            }

            int numTimeSteps = 0; 
            for (int i = 0; i < fixLen; i++) {
                numTimeSteps += fixTime[i] / timeStep; 
            }
            numTimeSteps++; 

            // requires compute capability 2.x
            float* barrierUp = new float[numTimeSteps];
            float *barrierDown = new float[numTimeSteps];

            for (int i = 0 ; i < numTimeSteps; i++) {
                barrierUp[i] = barrier / (1 + (dec * i));
                barrierDown[i] = -barrier / (1 + (dec * i));
            }

            double *prStates = new double[numStates];
            for (int i = 0; i < numStates; i++) {
                prStates[i] = (i == biasState) ? 1 : 0; 
            }

            double *probUpCrossing = new double[numTimeSteps];
            double *probDownCrossing = new double[numTimeSteps];
            for (int i = 0; i < numTimeSteps; i++) {
                probUpCrossing[i] = 0; 
                probDownCrossing[i] = 0; 
            }

            if (debug) {
                for (int i = 0 ; i < numStates ; i++) {
                    printf("prStates[%i] = %f\n", i, prStates[i]);
                }
            }

            int time = 1;

            float *changeMatrix = new float[numStates * numStates];
            for (int i = 0; i < numStates; i++) {
                for (int j = 0; j < numStates; j++) {
                    changeMatrix[__RC2IDX(i, j, numStates)] = states[i] - states[j];
                }
            }

            float *changeUp = new float[numStates * numTimeSteps];
            for (int i = 0; i < numStates; i++) {
                for (int j = 0; j < numTimeSteps; j++) {
                    changeUp[__RC2IDX(i, j, numTimeSteps)] = barrierUp[j] - states[i];
                }
            }

            float *changeDown = new float[numStates * numTimeSteps];
            for (int i = 0; i < numStates; i++) {
                for (int j = 0; j < numTimeSteps; j++) {
                    changeDown[__RC2IDX(i, j, numTimeSteps)] = barrierDown[j] - states[i];
                }
            }

            float *probDistChangeMatrix = new float[numStates * numStates];
            double* prStatesNew = new double[numStates];
            float *changeUpCDFs = new float[numStates];
            float *changeDownCDFs = new float[numStates];

            for (int f = 0; f < fixLen; f++) {
                int fItem = fixItem[f];
                int fTime = fixTime[f];

                if (debug) {
                    printf("fItem : %i ========== fTime : %i\n", fItem, fTime);
                }

                float mean; 
                if (fItem == 1) {
                    mean = d * (valLeft - (theta * valRight));
                } else if (fItem == 2) {
                    mean = d * ((theta * valLeft) - valRight);
                } else {
                    mean = 0; 
                }

                for (int i = 0; i < numStates; i++) {
                    for (int j = 0; j < numStates; j++) {
                        float x = changeMatrix[__RC2IDX(i, j, numStates)];
                        probDistChangeMatrix[__RC2IDX(i, j, numStates)] = pdf(x, mean, sigma);
                    }
                }
                if (debug) {
                    printf("PDCM\n");
                    for (int i = 0; i < numStates * numStates; i++) {
                        printf("%f ", probDistChangeMatrix[i]);
                        if ((i + 1) % numStates == 0) {
                            printf("\n");
                        }
                    }
                }

                for (int t = 0; t < fTime / timeStep; t++) {
                    double rowSum; 
                    for (int i = 0; i < numStates; i++) {
                        rowSum = 0; 
                        for (int j = 0; j < numStates; j++) {
                            rowSum += stateStep * probDistChangeMatrix[__RC2IDX(i, j, numStates)] * prStates[j];
                        }
                        prStatesNew[i] = (states[i] > barrierUp[time] || states[i] < barrierDown[time]) ? 0 : rowSum;
                    }

                    if (debug) {
                        for (int i = 0 ; i < numStates ; i++) {
                            printf("prStatesNew[%i] = %f\n", i, prStatesNew[i]);
                        }
                    }

                    for (int i = 0; i < numStates; i++) {
                        float x = changeUp[__RC2IDX(i, time, numTimeSteps)];
                        changeUpCDFs[i] = 1 - normcdff((x - mean) / sigma);
                    }
                    if (debug) {
                        for (int i = 0; i < numStates; i++) {
                            printf("changeUpCDFs[%i] = %f\n", i, changeUpCDFs[i]);
                        }
                    }
                    double tempUpCross = 0; 
                    for (int i = 0; i < numStates; i++) {
                        tempUpCross += changeUpCDFs[i] * prStates[i];
                    }

                    for (int i = 0; i < numStates; i++) {
                        float x = changeDown[__RC2IDX(i, time, numTimeSteps)];
                        changeDownCDFs[i] = normcdff((x - mean) / sigma);
                    }
                    if (debug) {
                        for (int i = 0; i < numStates; i++) {
                            printf("changeDownCDFs[%i] = %f\n", i, changeDownCDFs[i]);
                        }
                    }
                    double tempDownCross = 0; 
                    for (int i = 0; i < numStates; i++) {
                        tempDownCross += changeDownCDFs[i] * prStates[i];
                    }

                    if (debug) printf("temp up cross = %f\n", tempUpCross);
                    if (debug) printf("temp down cross = %f\n", tempDownCross);

                    double sumIn = 0; 
                    double sumCurrent = tempUpCross + tempDownCross; 
                    for (int i = 0; i < numStates; i++) {
                        sumIn += prStates[i];
                        sumCurrent += prStatesNew[i];
                    }
                    double normFactor = sumIn / sumCurrent; 
                    for (int i = 0; i < numStates; i++) {
                        prStates[i] = prStatesNew[i] * normFactor; 
                    }

                    probUpCrossing[time] = tempUpCross * normFactor; 
                    probDownCrossing[time] = tempDownCross * normFactor;

                    time++;
                }
            }

            double likelihood = 0; 
            if (choice == -1) {
                if (probUpCrossing[numTimeSteps - 1] > 0) {
                    likelihood = probUpCrossing[numTimeSteps - 1];
                }
            } else if (choice == 1) {
                if (probDownCrossing[numTimeSteps - 1] > 0) {
                    likelihood = probDownCrossing[numTimeSteps - 1];
                }
            }

            delete[] fixItem; 
            delete[] fixTime; 
            delete[] barrierUp;
            delete[] barrierDown;
            delete[] probUpCrossing;
            delete[] probDownCrossing;
            delete[] prStates;
            delete[] changeMatrix;
            delete[] changeUp;
            delete[] changeDown;
            delete[] probDistChangeMatrix;
            delete[] prStatesNew;
            delete[] changeUpCDFs;
            delete[] changeDownCDFs;
            
            likelihoods[trialNum] = -log(likelihood);            
        }
    }
}


void aDDM::callGetTrialLikelihoodKernel(
    bool debug, 
    int trialsPerThread,
    int numBlocks,
    int threadsPerBlock, 
    aDDMTrial *trials, 
    double *likelihoods, 
    int numTrials, 
    float d, 
    float sigma, 
    float theta, 
    float barrier, 
    int nonDecisionTime, 
    int timeStep, 
    float approxStateStep, 
    float decay
) {
    int *h_fixLens = new int[numTrials];
    int maxFixLen = 0; 
    for (int i = 0; i < numTrials; i++) {
        aDDMTrial trial = trials[i];
        assert(trial.fixItem.size() == trial.fixTime.size());
        int fixLen = trial.fixItem.size();
        if (fixLen > maxFixLen) {
            maxFixLen = fixLen;
        }
        h_fixLens[i] = fixLen; 
    }

    if (debug) std::cout << "max fix len " << maxFixLen << std::endl; 


    int *h_FIs = new int[numTrials * maxFixLen];
    int *h_FTs = new int[numTrials * maxFixLen];
    for (int i = 0; i < numTrials; i++) {
        for (int j = 0; j < maxFixLen; j++) {
            aDDMTrial trial = trials[i];
            if (j < trial.fixItem.size()) {
                h_FIs[__RC2IDX(i, j, maxFixLen)] = trial.fixItem[j]; 
                h_FTs[__RC2IDX(i, j, maxFixLen)] = trial.fixTime[j]; 
            } else {
                h_FIs[__RC2IDX(i, j, maxFixLen)] = -1; 
                h_FTs[__RC2IDX(i, j, maxFixLen)] = -1; 
            }
        }
    }

    if (debug) {
        for (int i = 0; i < numTrials; i++) {
            std::cout << "[" << h_fixLens[i] << "] ";
            for (int j = 0; j < maxFixLen; j++) {
                std::cout << h_FIs[__RC2IDX(i, j, maxFixLen)] << " ";  
            }
            std::cout << std::endl;
        }
    }

    int *h_VLs = new int[numTrials];
    int *h_VRs = new int[numTrials];
    int *h_RTs = new int[numTrials];
    int *h_choices = new int[numTrials];
    for (int i = 0; i < numTrials; i++) {
        aDDMTrial trial = trials[i];
        h_VLs[i] = trial.valueLeft; 
        h_VRs[i] = trial.valueRight; 
        h_RTs[i] = trial.RT;
        h_choices[i] = trial.choice; 
    }

    int *d_RTs, *d_choices, *d_VLs, *d_VRs, *d_FIs, *d_FTs, *d_FixLens;
    hipMalloc((void **) &d_RTs, numTrials * sizeof(int));
    hipMalloc((void **) &d_choices, numTrials * sizeof(int));
    hipMalloc((void **) &d_VLs, numTrials * sizeof(int));
    hipMalloc((void **) &d_VRs, numTrials * sizeof(int));
    hipMalloc((void **) &d_FIs, numTrials * maxFixLen * sizeof(int));
    hipMalloc((void **) &d_FTs, numTrials * maxFixLen * sizeof(int)); 
    hipMalloc((void **) &d_FixLens, numTrials * sizeof(int));

    hipMemcpy(d_RTs, h_RTs, numTrials * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_choices, h_choices, numTrials * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_VLs, h_VLs, numTrials * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_VRs, h_VRs, numTrials * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_FIs, h_FIs, numTrials * maxFixLen * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_FTs, h_FTs, numTrials * maxFixLen * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_FixLens, h_fixLens, numTrials * sizeof(int), hipMemcpyHostToDevice);

    int halfNumStateBins = ceil(barrier / approxStateStep); 
    if (debug) printf("half num state bins %i\n", halfNumStateBins);
    float stateStep = barrier / (halfNumStateBins + 0.5);
    if (debug) printf("state step %f\n", stateStep);
    int numStates = 2 * halfNumStateBins + 1; 

    float *states = new float[numStates];
    int s_idx = 0; 
    float biasStateVal = MAXFLOAT; 
    int biasState; 
    float r; 
    
    for (float ss = -barrier + (stateStep / 2); ss <= barrier - (stateStep / 2); ss += stateStep) {
        states[s_idx] = ss;
        r = abs(ss - bias); 
        if (r < biasStateVal) {
            biasState = s_idx;
            biasStateVal = r; 
        }
        s_idx++;
    }

    float *d_states; 
    hipMalloc((void**) &d_states, numStates * sizeof(float));
    hipMemcpy(d_states, states, numStates * sizeof(float), hipMemcpyHostToDevice);

    getTrialLikelihoodKernel<<<numBlocks, threadsPerBlock>>>(
        debug, 
        trialsPerThread, 
        d_RTs, 
        d_choices, 
        d_VLs, 
        d_VRs, 
        d_FIs, 
        d_FTs, 
        d_FixLens, 
        likelihoods, 
        numTrials, 
        d_states, 
        maxFixLen,
        biasState, 
        numStates,
        stateStep,
        d, sigma, theta, barrier, 
        nonDecisionTime, 
        timeStep, 
        approxStateStep, 
        decay
    );

    hipFree(d_RTs);
    hipFree(d_choices);
    hipFree(d_VLs);
    hipFree(d_VRs);
    hipFree(d_FIs);
    hipFree(d_FTs);
    hipFree(d_FixLens);
    delete[] h_RTs;
    delete[] h_choices; 
    delete[] h_VLs;
    delete[] h_VRs;
    delete[] h_FIs;
    delete[] h_FTs;
    delete[] h_fixLens;
}


double aDDM::computeGPUNLL(std::vector<aDDMTrial> trials, bool debug, int trialsPerThread, int timeStep, float approxStateStep) {
    int numTrials = trials.size();

    aDDMTrial* d_trials;
    double *d_likelihoods;
    hipMalloc((void **) &d_trials, numTrials * sizeof(aDDMTrial));
    hipMalloc((void **) &d_likelihoods, numTrials * sizeof(double));
    hipMemcpy(d_trials, trials.data(), numTrials * sizeof(aDDMTrial), hipMemcpyHostToDevice);

    int threadsPerBlock = 256; 
    int numBlocks = 16; 

    aDDM::callGetTrialLikelihoodKernel(
        debug, trialsPerThread, numBlocks, threadsPerBlock,
        trials.data(), d_likelihoods, numTrials, 
        d, sigma, theta, barrier, 
        nonDecisionTime, timeStep, approxStateStep, DECAY
    );

    std::vector<double> h_likelihoods(numTrials);
    hipMemcpy(h_likelihoods.data(), d_likelihoods, numTrials * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_trials);
    hipFree(d_likelihoods);

    double NLL = 0;
    for (int i = 0; i < numTrials; i++) {
        NLL += h_likelihoods[i];
    }

    return NLL;
}

aDDM aDDM::fitModelMLE(std::vector<aDDMTrial> trials, std::vector<float> rangeD, std::vector<float> rangeSigma, std::vector<float> rangeTheta, float barrier, std::string computeMethod) {
    if (std::find(validComputeMethods.begin(), validComputeMethods.end(), computeMethod) == validComputeMethods.end()) {
        throw std::invalid_argument("Input computeMethod is invalid.");
    }

    std::vector<aDDM> potentialModels; 
    for (float d : rangeD) {
        for (float sigma : rangeSigma) {
            for (float theta : rangeTheta) {
                aDDM addm = aDDM(d, sigma, theta, barrier);
                potentialModels.push_back(addm);
            }
        }
    }

    std::function<double(aDDM)> NLLcomputer; 
    if (computeMethod == "basic") {
        NLLcomputer = [trials](aDDM addm) -> double {
            double NLL = 0; 
            for (aDDMTrial trial : trials) {
                NLL += -log(addm.getTrialLikelihood(trial));
            }
            return NLL; 
        };
    }
    else if (computeMethod == "thread") {
        NLLcomputer = [trials](aDDM addm) -> double {
            return addm.computeParallelNLL(trials);
        };
    }
    else if (computeMethod == "gpu") {
        NLLcomputer = [trials](aDDM addm) -> double {
            return addm.computeGPUNLL(trials);
        };
    }

    double minNLL = __DBL_MAX__; 
    aDDM optimal = aDDM(); 
    for (aDDM addm : potentialModels) {
        std::cout << "testing d=" << addm.d << " sigma=" << addm.sigma << " theta=" << addm.theta << std::endl; 
        double NLL = NLLcomputer(addm);
        if (NLL < minNLL) {
            minNLL = NLL; 
            optimal = addm; 
        }
    }
    return optimal; 

}

