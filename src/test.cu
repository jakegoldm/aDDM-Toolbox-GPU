#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <fstream>
#include <stdio.h>
#include <sstream>
#include <iomanip>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/extrema.h>
#include <thrust/reduce.h>
#include <thrust/inner_product.h>
#include "../include/ddm.h"
#include "../include/util.h"


std::vector<float> rangeD = {0.005};
std::vector<float> rangeSigma = {0.07};

int barrier = 1;
int valueLeft = 3; 

int timeStep = 10;
float approxStateStep = 0.1; 


struct abs_diff_functor
{
    const float bias;

    __host__ __device__
    float operator()(const float& x) const
    {
        return fabs(x - bias);
    }
};

struct cdf_functor
{
    float a;
    float b; 
    float m; 
    float s;

    cdf_functor(float _a, float _b, float _m, float _s) : a(_a), b(_b), m(_m), s(_s) {}

    __host__ __device__ 
    float operator()(const float& x) const 
    {
        return a + b * normcdf((x - m) / s);
    }
};

__device__ double devicePDF(float x, float mean, float sigma) {
    float first = exp(-0.5 * pow((x - mean) / sigma, 2));
    float second = sigma * sqrt(2 * M_PI);
    return first / second;
}

__global__ void computeChangeMatrix(const float* states, float* changeMatrix, size_t numStates) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < numStates && j < numStates) {
        int index = i * numStates + j;
        changeMatrix[index] = states[j] - states[i];
    }
}

__global__ void computeChange(const float* barrier, const float* states, float* changeMat, size_t numStates, size_t numTimeSteps) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < numStates && j < numTimeSteps) {
        int index = j * numStates + i;
        changeMat[index] = barrier[j] - states[i];
    }
}

__global__ void computeProbDistChangeMatrix(const float* changeMatrix, double *probDistChangeMatrix, size_t numStates, float mean, float sigma) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < numStates && j < numStates) {
        int index = j * numStates + i;
        float x = changeMatrix[index];
        double y = devicePDF(x, mean, sigma);
        probDistChangeMatrix[index] = y;
    }    
}

__global__ void computePrStatesNew(const double* probDistChangeMatrix, const double* prevTimeSlice, double* prStatesNew, size_t numStates, float stateStep) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < numStates) {
        double dotProduct = 0.0;
        for (int j = 0; j < numStates; j++) {
            int index = j * numStates + i;
            dotProduct += probDistChangeMatrix[index] * prevTimeSlice[j];
        }
        prStatesNew[i] = dotProduct * stateStep;
    }
}




double getTrialLikelihoodGPU(DDM ddm, DDMTrial trial) {

    int timeStep = 10;
    float approxStateStep = 0.1;

    int numTimeSteps = trial.RT / timeStep;

    std::cout << std::setprecision(6) << std::fixed; 

    // std::cout << "RT = " << trial.RT << std::endl; 

    thrust::device_vector<float> barrierUp(numTimeSteps, ddm.barrier);
    thrust::device_vector<float> barrierDown(numTimeSteps, -ddm.barrier);

    int halfNumStateBins = ceil(ddm.barrier / approxStateStep);
    float stateStep = ddm.barrier / (halfNumStateBins + 0.5);

    thrust::device_vector<float> states(1 + (barrierUp[0] - barrierDown[0] - (stateStep / 2)) / stateStep);
    thrust::sequence(states.begin(), states.end(), barrierDown[0] + stateStep / 2, stateStep);

    // for (int i = 0; i < states.size(); i++) {
    //     std::cout << "B[" << i << "] = " << states[i] << std::endl;
    // }

    int numStates = states.size();
    thrust::device_vector<float> differences(numStates);
    thrust::transform(states.begin(), states.end(), differences.begin(), thrust::placeholders::_1 - ddm.bias);
    thrust::transform(differences.begin(), differences.end(), differences.begin(), abs_diff_functor{ddm.bias});
    auto min_diff_iter = thrust::min_element(differences.begin(), differences.end());
    int bias_idx = min_diff_iter - differences.begin();

    dim3 blockSize(16, 16);  // Adjust the block size according to your GPU architecture
    dim3 gridSize((numStates + blockSize.x - 1) / blockSize.x, (numStates + blockSize.y - 1) / blockSize.y);

    // std::cout << "bias index " << bias_idx << std::endl;

    // Column Major Ordering 
    // prStates[state][time] = prStates[time * numStates + state]
    // flattened matrix: all timesteps for a given state sequentially
    thrust::device_vector<double> prStates(numStates * numTimeSteps, 0); 
    prStates[bias_idx] = 1;

    thrust::device_vector<double> probUpCrossing(numTimeSteps, 0);
    thrust::device_vector<double> probDownCrossing(numTimeSteps, 0);

    // changeMatrix[i][j] = changeMatrix[i * numStates + j]
    thrust::device_vector<float> changeMatrix(numStates * numStates);

    thrust::device_vector<float> changeUp(numStates * numTimeSteps);    
    thrust::device_vector<float> changeDown(numStates * numTimeSteps);
    for (size_t i = 0; i < numStates; i++) {
        for (size_t j = 0; j < numTimeSteps; j++) {
            changeUp[j * numStates + i] = barrierUp[j] - states[i];
        }
    }
    for (size_t i = 0; i < numStates; i++) {
        for (size_t j = 0; j < numTimeSteps; j++) {
            changeDown[j * numStates + i] = barrierDown[j] - states[i];
        }
    }

    computeChangeMatrix<<<gridSize, blockSize>>>(
        thrust::raw_pointer_cast(states.data()), 
        thrust::raw_pointer_cast(changeMatrix.data()), 
        numStates);
    computeChange<<<gridSize, blockSize>>>(
        thrust::raw_pointer_cast(barrierUp.data()), 
        thrust::raw_pointer_cast(states.data()), 
        thrust::raw_pointer_cast(changeUp.data()), 
        numStates, numTimeSteps);
    computeChange<<<gridSize, blockSize>>>(
        thrust::raw_pointer_cast(barrierDown.data()), 
        thrust::raw_pointer_cast(states.data()), 
        thrust::raw_pointer_cast(changeDown.data()), 
        numStates, numTimeSteps);

    std::cout << "CHANGE MATRIX" << std::endl; 
    for (int i = 0; i < changeMatrix.size(); i++) {
        std::cout << changeMatrix[i] << " "; 
        if ((i + 1) % numStates == 0) { std::cout << std::endl; }
    }
    std::cout << "CHANGE UP" << std::endl; 
    for (int i = 0; i < changeUp.size(); i++) {
        std::cout << changeUp[i] << " "; 
        if ((i + 1) % numStates == 0) { std::cout << std::endl; }
    }
    std::cout << "CHANGE DOWN" << std::endl; 
    for (int i = 0; i < changeDown.size(); i++) {
        std::cout << changeDown[i] << " "; 
        if ((i + 1) % numStates == 0) { std::cout << std::endl; }
    }

    int elapsedNDT = 0; 
    for (int time = 1; time < numTimeSteps; time++) {
        std::cout << "============" << std::endl;
        std::cout << "TIMESTEP " << time << std::endl;
        std::cout << "============" << std::endl;        


        float mean; 
        if (elapsedNDT < ddm.nonDecisionTime / timeStep) {
            mean = 0; 
            elapsedNDT += 1; 
        } else {
            mean = ddm.d * (trial.valueLeft - trial.valueRight);
        }

        thrust::device_vector<double> probDistChangeMatrix(numStates * numStates);
        computeProbDistChangeMatrix<<<gridSize, blockSize>>>(
            thrust::raw_pointer_cast(changeMatrix.data()), 
            thrust::raw_pointer_cast(probDistChangeMatrix.data()), 
            numStates, mean, ddm.sigma
        );

        std::cout << "PROB DIST CHANGE" << std::endl; 
        for (int i = 0; i < probDistChangeMatrix.size(); i++) {
            std::cout << probDistChangeMatrix[i] << " "; 
            if ((i + 1) % numStates == 0) { std::cout << std::endl; }
        }

        thrust::device_vector<double> prevTimeSlice(numStates);
        thrust::copy(
            prStates.begin() + (time - 1) * numStates, 
            prStates.begin() + (time) * numStates, 
            prevTimeSlice.begin()
        );

        std::cout << "PREV TIME SLICE" << std::endl; 
        for (double f : prevTimeSlice) { 
            std::cout << f << std::endl; 
        }
        std::cout << "numStates " << numStates << std::endl; ;
        std::cout << "time slice " << prevTimeSlice.size() << std::endl; 

        thrust::device_vector<double> prStatesNew(numStates);

        computePrStatesNew<<<gridSize, blockSize>>>(
            thrust::raw_pointer_cast(probDistChangeMatrix.data()), 
            thrust::raw_pointer_cast(prevTimeSlice.data()), 
            thrust::raw_pointer_cast(prStatesNew.data()), 
            numStates, stateStep
        );
        std::cout << "PR STATES NEW" << std::endl; 
        for (double d : prStatesNew) {
            std::cout << d << std::endl; 
        }

        thrust::device_vector<float> currChangeUp(numStates); 
        thrust::transform(
            changeUp.begin() + (time) * numTimeSteps, 
            changeUp.begin() + (time + 1) * numTimeSteps, 
            currChangeUp.begin(), 
            cdf_functor(1, -1, mean, ddm.sigma)
        );
        std::cout << "CURR CHANGE UP (CDF)" << std::endl; 
        for (float f : currChangeUp) {
            std::cout << f << std::endl; 
        }

        double tempUpCross = 0; 
        for (int i = 0; i < numStates; i++) {
            tempUpCross += currChangeUp[i] * prevTimeSlice[i];
        }
        // double tempUpCross = thrust::inner_product(currChangeUp.begin(), currChangeUp.end(), prevTimeSlice.begin(), 0);
        std::cout << "temp up cross " << tempUpCross << std::endl; 

        thrust::device_vector<float> currChangeDown(numStates);
        thrust::transform(
            changeDown.begin() + (time) * numTimeSteps, 
            changeDown.begin() + (time + 1) * numTimeSteps, 
            currChangeDown.begin(),
            cdf_functor(0, 1, mean, ddm.sigma)
        );
        std::cout << "CURR CHANGE DOWN (CDF)" << std::endl; 
        for (float f : currChangeDown) { 
            std::cout << f << std::endl; 
        }

        double tempDownCross = 0; 
        for (int i = 0; i < numStates; i++) {
            tempDownCross += currChangeDown[i] * prevTimeSlice[i];
        }
        // double tempDownCross = thrust::inner_product(currChangeDown.begin(), currChangeDown.end(), prevTimeSlice.begin(), 0);
        std::cout << "temp down cross " << tempDownCross << std::endl; 

        double sumIn = thrust::reduce(prevTimeSlice.begin(), prevTimeSlice.end(), 0, thrust::plus<double>());
        double sumCurrent = thrust::reduce(
            prStatesNew.begin(), prStatesNew.end(), 0, thrust::plus<double>()
            ) + tempUpCross + tempDownCross;
        double normFactor = sumIn / sumCurrent; 
        tempUpCross *= normFactor; 
        tempDownCross *= normFactor; 

        thrust::transform(prStatesNew.begin(), prStatesNew.end(), prStatesNew.begin(), thrust::placeholders::_1 * normFactor);
        thrust::copy(prStatesNew.begin(), prStatesNew.end(), prStates.begin() + (time) * numStates);
        
        std::cout << "UPDATED PRSTATES" << std::endl; 
        for (int i = 0; i < prStates.size(); i++) {
            std::cout << prStates[i] << " "; 
            if ((i + 1) % numStates == 0) { std::cout << std::endl; }
        }

        for (int i = 0; i < 200; i++) {
            std::cout << "=";
        }
        std::cout << std::endl; 


        probUpCrossing[time] = tempUpCross;
        probDownCrossing[time] = tempDownCross;
    }
    double likelihood = 0; 
    if (trial.choice == -1) {
        if (probUpCrossing[numTimeSteps - 1] > 0) {
            likelihood = probUpCrossing[numTimeSteps - 1];
        }
    }
    else if (trial.choice == 1) {
        if (probDownCrossing[numTimeSteps - 1] > 0) {
            likelihood = probDownCrossing[numTimeSteps - 1];
        }
    }
    return likelihood;
}

int main() {

    std::vector<DDMTrial> trials;
    std::vector<DDM> ddms;
    std::ifstream file("results/ddm_simulations.csv");
    std::string line;
    std::getline(file, line);
    int choice;
    int RT;
    int valDiff;
    while (std::getline(file, line)) {
        std::stringstream ss(line);
        std::string field;
        std::getline(ss, field, ',');
        choice = std::stoi(field);
        std::getline(ss, field, ',');
        RT = std::stoi(field);
        std::getline(ss, field, ',');
        valDiff = std::stoi(field);
        DDMTrial dt = DDMTrial(RT, choice, valueLeft, valueLeft - valDiff);
        trials.push_back(dt);
    }
    file.close();
    std::cout << "Counted " << trials.size() << " trials." << std::endl;

    for (float d : rangeD) {
        for (float sigma : rangeSigma) {
            ddms.push_back(DDM(d, sigma, barrier));
        }
    }


    for (DDM& ddm : ddms) {
        double NLL = 0; 
        for (DDMTrial& trial : trials) {

            double prob = getTrialLikelihoodGPU(ddm, trial);

            break;
        }
        std::cout << "NLL: " << NLL << std::endl;
    }

    return 0;
}